#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *                  Yuvaraj Chesetti <chesetti@cs.utah.edu>
 *                  Ashish Tiwari <hi@aashishtiwari.com.np>
 *
 * ============================================================================
 */





#include <gallatin/allocators/global_allocator.cuh>

#include <gallatin/allocators/timer.cuh>

#include <gpu_pointers/dummy_ptr.cuh>


#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <chrono>
#include <openssl/rand.h>



// #include <hashing_project/table_wrappers/p2_wrapper.cuh>
// #include <hashing_project/table_wrappers/dummy_ht.cuh>
// #include <hashing_project/table_wrappers/iht_wrapper.cuh>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

using namespace gallatin::allocators;

#define CHECK_CORRECTNESS 1


#if GALLATIN_DEBUG_PRINTS
   #define TEST_BLOCK_SIZE 256
#else
   #define TEST_BLOCK_SIZE 256
#endif


template <typename T>
__host__ T * generate_data(uint64_t nitems){


   //malloc space

   T * vals;

   hipHostMalloc((void **)&vals, sizeof(T)*nitems);


   //          100,000,000
   uint64_t cap = 100000000ULL;

   for (uint64_t to_fill = 0; to_fill < nitems; to_fill+=0){

      uint64_t togen = (nitems - to_fill > cap) ? cap : nitems - to_fill;


      RAND_bytes((unsigned char *) (vals + to_fill), togen * sizeof(T));



      to_fill += togen;

      //printf("Generated %llu/%llu\n", to_fill, nitems);

   }

   printf("Generation done\n");
   return vals;
}

template <template<typename> typename pointer_type, typename T>
__global__ void test_add_kernel(pointer_type<T> * ptr, uint64_t * bitarray, uint64_t n_ops){


   uint64_t tid = gallatin::utils::get_tid();

   if (tid >= n_ops) return;

   T my_value = 1;

   T old = ptr->atomicAdd(my_value);


   #if CHECK_CORRECTNESS

   uint64_t high = old/64;
   uint64_t low = old % 64;


   if (atomicOr((unsigned long long int *)&bitarray[high], (unsigned long long int) SET_BIT_MASK(low)) & SET_BIT_MASK(low)){
      printf("Double add to index %llu\n", old);
   }


   #endif


}


template <template<typename> typename pointer_type, typename T>
__global__ void test_exch_kernel(pointer_type<T> * ptr, uint64_t * bitarray, uint64_t n_ops){


   uint64_t tid = gallatin::utils::get_tid();

   if (tid >= n_ops) return;

   T my_value = tid;

   T old = ptr->atomicExch(my_value);


   #if CHECK_CORRECTNESS

   uint64_t high = old/64;
   uint64_t low = old % 64;


   if (atomicOr((unsigned long long int *)&bitarray[high], (unsigned long long int) SET_BIT_MASK(low)) & SET_BIT_MASK(low)){
      printf("Double add to index %llu\n", old);
   }


   #endif

}

// template <typename pointer_type, typename T>
// __global__ void test_cas_kernel(pointer_type * ptr, uint64_t * bitarray, uint64_t n_ops){


//    uint64_t tid = gallatin::utils::get_tid();

//    if (tid >= n_ops) return;

//    T my_value = tid;

//    T current_value = ptr->load_acq();

//    while (true){

//       T next_value = ptr->atomicCAS(current_value, my_value)

//       if (next_value == current_value) break;

//       __threadfence();
//       current_value = next_value;
      
//    }


//    #if CHECK_CORRECTNESS

//    uint64_t high = current_value/64;
//    uint64_t low = current_value % 64;


//    if (atomicOr((unsigned long long int *)&bitarray[high], (unsigned long long int) SET_BIT_MASK(low)) & SET_BIT_MASK(low)){
//       printf("Double add to index %llu\n", current_value);
//    }


//    #endif


// }


//pull from blocks
//this kernel tests correctness, and outputs misses in a counter.
//works on actual pointers instead of uint64_t
//The correctness check is done by treating each allocation as a uint64_t and writing the tid
// if TID is not what is expected, we know that a double malloc has occurred.
template <template<typename> typename pointer_type>
__host__ void ptr_add_test(uint64_t n_ops){


   using ptr_type = pointer_type<uint64_t>;

   ptr_type * dev_ptr = ptr_type::generate_on_device(0ULL);

   uint64_t n_lock_uints = (n_ops)/64+1;

   uint64_t * bitarray = gallatin::utils::get_device_version<uint64_t>(n_lock_uints);

   hipMemset(bitarray, 0ULL, sizeof(uint64_t)*n_lock_uints);




   gallatin::utils::timer add_timer;

   test_add_kernel<pointer_type, uint64_t><<<(n_ops-1)/1024+1,1024>>>(dev_ptr, bitarray, n_ops);

   add_timer.sync_end();

   add_timer.print_throughput("Added", n_ops);


   ptr_type::free_on_device(dev_ptr);
   hipFree(bitarray);

   //hipFree(access_data);

}


template <template<typename> typename pointer_type>
__host__ void ptr_exch_test(uint64_t n_ops){


   using ptr_type = pointer_type<uint64_t>;

   ptr_type * dev_ptr = ptr_type::generate_on_device(n_ops);

   uint64_t n_lock_uints = (n_ops)/64+1;

   uint64_t * bitarray = gallatin::utils::get_device_version<uint64_t>(n_lock_uints);

   hipMemset(bitarray, 0ULL, sizeof(uint64_t)*n_lock_uints);




   gallatin::utils::timer add_timer;

   test_exch_kernel<pointer_type, uint64_t><<<(n_ops-1)/1024+1,1024>>>(dev_ptr, bitarray, n_ops);

   add_timer.sync_end();

   add_timer.print_throughput("Exchanged", n_ops);


   ptr_type::free_on_device(dev_ptr);
   hipFree(bitarray);

   //hipFree(access_data);

}


int main(int argc, char** argv) {

   uint64_t n_ops;



   if (argc < 2){
      n_ops = 1000000;
   } else {
      n_ops = std::stoull(argv[1]);
   }


   ptr_add_test<gpu_pointers::dummy_pointer>(n_ops);
   

   ptr_exch_test<gpu_pointers::dummy_pointer>(n_ops);


   hipDeviceReset();
   return 0;

}
