#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *                  Yuvaraj Chesetti <chesetti@cs.utah.edu>
 *                  Ashish Tiwari <hi@aashishtiwari.com.np>
 *
 * ============================================================================
 */





#include <gallatin/allocators/global_allocator.cuh>

#include <gallatin/allocators/timer.cuh>

#include <gpu_pointers/dummy_ptr.cuh>
#include <gpu_pointers/coalesce_ptr.cuh>


#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <chrono>
#include <openssl/rand.h>



// #include <hashing_project/table_wrappers/p2_wrapper.cuh>
// #include <hashing_project/table_wrappers/dummy_ht.cuh>
// #include <hashing_project/table_wrappers/iht_wrapper.cuh>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

using namespace gallatin::allocators;

#define CHECK_CORRECTNESS 0


#if GALLATIN_DEBUG_PRINTS
   #define TEST_BLOCK_SIZE 256
#else
   #define TEST_BLOCK_SIZE 256
#endif


template <typename T>
__host__ T * generate_data(uint64_t nitems){


   //malloc space

   T * vals;

   hipHostMalloc((void **)&vals, sizeof(T)*nitems);


   //          100,000,000
   uint64_t cap = 100000000ULL;

   for (uint64_t to_fill = 0; to_fill < nitems; to_fill+=0){

      uint64_t togen = (nitems - to_fill > cap) ? cap : nitems - to_fill;


      RAND_bytes((unsigned char *) (vals + to_fill), togen * sizeof(T));



      to_fill += togen;

      //printf("Generated %llu/%llu\n", to_fill, nitems);

   }

   printf("Generation done\n");
   return vals;
}

template <template<typename> typename pointer_type, typename T>
__global__ void test_add_kernel(pointer_type<T> * ptr, pointer_type<T> * alt_ptr, uint64_t * bitarray, uint64_t n_ops){


   uint64_t tid = gallatin::utils::get_tid();

   if (tid >= n_ops) return;

   T my_value = 1;

   T old;

   if (tid % 2 == 0){
      old = ptr->atomicAdd(my_value);
   } else {
      old = alt_ptr->atomicAdd(my_value);
   }


}


template <template<typename> typename pointer_type, typename T>
__global__ void test_exch_kernel(pointer_type<T> * ptr, pointer_type<T> * alt_ptr, uint64_t * bitarray, uint64_t n_ops){


   uint64_t tid = gallatin::utils::get_tid();

   if (tid >= n_ops) return;

   T my_value = tid;

   T old;

   if (tid % 2 == 0){
      old  = ptr->atomicExch(my_value);
   } else {
      old  = alt_ptr->atomicExch(my_value);
   }

}

template <template<typename> typename pointer_type, typename T>
__global__ void test_rmw_kernel(pointer_type<T> * ptr, pointer_type<T> * alt_ptr, uint64_t * bitarray, uint64_t n_ops){


   uint64_t tid = gallatin::utils::get_tid();

   if (tid >= n_ops) return;

   
   auto add_lambda = [](T a) { uint64_t ret_val = a+1; return ret_val;};

   T old;

   if (tid % 2 == 0){
      old  = ptr->apply_rmw(add_lambda);
   } else {
      old  = alt_ptr->apply_rmw(add_lambda);
   }


   //printf("Done with %lu\n", tid);

}

// template <typename pointer_type, typename T>
// __global__ void test_cas_kernel(pointer_type * ptr, uint64_t * bitarray, uint64_t n_ops){


//    uint64_t tid = gallatin::utils::get_tid();

//    if (tid >= n_ops) return;

//    T my_value = tid;

//    T current_value = ptr->load_acq();

//    while (true){

//       T next_value = ptr->atomicCAS(current_value, my_value)

//       if (next_value == current_value) break;

//       __threadfence();
//       current_value = next_value;
      
//    }


//    #if CHECK_CORRECTNESS

//    uint64_t high = current_value/64;
//    uint64_t low = current_value % 64;


//    if (atomicOr((unsigned long long int *)&bitarray[high], (unsigned long long int) SET_BIT_MASK(low)) & SET_BIT_MASK(low)){
//       printf("Double add to index %llu\n", current_value);
//    }


//    #endif


// }


//pull from blocks
//this kernel tests correctness, and outputs misses in a counter.
//works on actual pointers instead of uint64_t
//The correctness check is done by treating each allocation as a uint64_t and writing the tid
// if TID is not what is expected, we know that a double malloc has occurred.
template <template<typename> typename pointer_type>
__host__ void ptr_add_test(uint64_t n_ops){


   using ptr_type = pointer_type<uint64_t>;

   ptr_type * dev_ptr = ptr_type::generate_on_device(0ULL);

   ptr_type * alt_ptr = ptr_type::generate_on_device(0ULL);

   uint64_t n_lock_uints = (n_ops)/64+1;

   uint64_t * bitarray = gallatin::utils::get_device_version<uint64_t>(n_lock_uints);

   hipMemset(bitarray, 0ULL, sizeof(uint64_t)*n_lock_uints);




   gallatin::utils::timer add_timer;

   test_add_kernel<pointer_type, uint64_t><<<(n_ops-1)/1024+1,1024>>>(dev_ptr, alt_ptr, bitarray, n_ops);

   add_timer.sync_end();

   add_timer.print_throughput("Added", n_ops);


   ptr_type::free_on_device(dev_ptr);
   ptr_type::free_on_device(alt_ptr);
   hipFree(bitarray);

   //hipFree(access_data);

}


template <template<typename> typename pointer_type>
__host__ void ptr_exch_test(uint64_t n_ops){


   using ptr_type = pointer_type<uint64_t>;

   ptr_type * dev_ptr = ptr_type::generate_on_device(n_ops);
   ptr_type * alt_ptr = ptr_type::generate_on_device(n_ops);

   uint64_t n_lock_uints = (n_ops)/64+1;

   uint64_t * bitarray = gallatin::utils::get_device_version<uint64_t>(n_lock_uints);

   hipMemset(bitarray, 0ULL, sizeof(uint64_t)*n_lock_uints);




   gallatin::utils::timer add_timer;

   test_exch_kernel<pointer_type, uint64_t><<<(n_ops-1)/1024+1,1024>>>(dev_ptr, alt_ptr, bitarray, n_ops);

   add_timer.sync_end();

   add_timer.print_throughput("Exchanged", n_ops);


   ptr_type::free_on_device(dev_ptr);
   ptr_type::free_on_device(alt_ptr);
   hipFree(bitarray);

   //hipFree(access_data);

}


template <template<typename> typename pointer_type>
__host__ void ptr_rmw_test(uint64_t n_ops){


   using ptr_type = pointer_type<uint64_t>;

   ptr_type * dev_ptr = ptr_type::generate_on_device(0ULL);
   ptr_type * alt_ptr = ptr_type::generate_on_device(0ULL);

   uint64_t n_lock_uints = (n_ops)/64+1;

   uint64_t * bitarray = gallatin::utils::get_device_version<uint64_t>(n_lock_uints);

   hipMemset(bitarray, 0ULL, sizeof(uint64_t)*n_lock_uints);




   gallatin::utils::timer add_timer;

   test_rmw_kernel<pointer_type, uint64_t><<<(n_ops-1)/1024+1,1024>>>(dev_ptr, alt_ptr, bitarray, n_ops);

   add_timer.sync_end();

   add_timer.print_throughput("RMW'ed", n_ops);


   ptr_type::free_on_device(dev_ptr);
   ptr_type::free_on_device(alt_ptr);
   hipFree(bitarray);

   //hipFree(access_data);

}

int main(int argc, char** argv) {

   uint64_t n_ops;



   if (argc < 2){
      n_ops = 1000000;
   } else {
      n_ops = std::stoull(argv[1]);
   }


   printf("Dummy ptr\n");
   ptr_add_test<gpu_pointers::dummy_pointer>(n_ops);

   ptr_exch_test<gpu_pointers::dummy_pointer>(n_ops);

   ptr_rmw_test<gpu_pointers::dummy_pointer>(1000000);


   printf("Coalesced ptr\n");
   ptr_add_test<gpu_pointers::coalesce_pointer>(n_ops);

   ptr_exch_test<gpu_pointers::coalesce_pointer>(n_ops);

   ptr_rmw_test<gpu_pointers::coalesce_pointer>(n_ops);


   hipDeviceReset();
   return 0;

}
